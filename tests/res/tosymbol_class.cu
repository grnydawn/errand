#include "stdio.h"
#include <iostream>
#include <hip/hip_runtime.h>



class A {
public:
	int x;
};

A * var;

__global__ void test(A * dvar){
	dvar->x = 1;
}

int main(int argc, char* argv[])
{
    //int num = 25;
    //cudaMemcpyToSymbol(dim,&num,sizeof(int),0,cudaMemcpyHostToDevice);
    
    //int *gpu_Num;
    hipMalloc(&var,sizeof(A));

    test<<<1,1>>>(var);

	int p;
    hipMemcpy(&p,&(var->x),sizeof(int),hipMemcpyDefault);
    
    printf("Result: %i\n",p);
}
