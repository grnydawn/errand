
#include <hip/hip_runtime.h>
void * devmalloc(size_t size) {
    void * d;

    hipMalloc(&d, size);

    return d;
}

void * memcpy2host(void * h, void * d, size_t size) {

	hipMemcpy(h, d, size, hipMemcpyDeviceToHost);

    return h;
}

void devfree(void * d) {

	hipFree(d);
}
