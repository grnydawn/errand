#include "stdio.h"
#include <iostream>
#include <hip/hip_runtime.h>


class A {
public:
	int * x;
};

A var = A();

__global__ void test(A & dvar){

    //dvar.x = y;

	dvar.x[0] = 10;
	dvar.x[1] = 20;
}

int main(int argc, char* argv[])
{
    hipMalloc(&(var.x),sizeof(int)*2);

    test<<<1,1>>>(var);

	int p[2];
    hipMemcpy(p,var.x,sizeof(int)*2,hipMemcpyDefault);
    
    printf("Result: %i, %i\n",p[0], p[1]);
}
