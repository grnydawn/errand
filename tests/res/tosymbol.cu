#include "stdio.h"
#include <iostream>
#include <hip/hip_runtime.h>


__constant__ int dim;

__global__ void test(int *gpu_Num){
    *gpu_Num = dim;
}

int main(int argc, char* argv[])
{
    int num = 25;
    hipMemcpyToSymbol(HIP_SYMBOL(dim),&num,sizeof(int),0,hipMemcpyHostToDevice);
    
    int *gpu_Num;
    hipMalloc(&gpu_Num,sizeof(int));

    test<<<1,1>>>(gpu_Num);

    int hostResult;
    hipMemcpy(&hostResult,gpu_Num,sizeof(int),hipMemcpyDefault);
    
    printf("Result: %i\n",hostResult);
}
